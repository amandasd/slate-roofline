#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <mpi.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#ifdef PAPI
#    include <papi.h>
#endif

#include "interface.h"

__global__ void
MatAdd(float* A, float* B, float* C, int n, int taskperItem)
{
    // Get our global thread ID
    int global_id = blockIdx.x * blockDim.x * taskperItem + threadIdx.x;
    for(int t = 0; t < taskperItem; t++)
    {
        int i = t * blockDim.x + global_id;
        // Make sure we do not go out of bounds
        if(i < n)
            C[i] = A[i] + B[i];
    }
}

int
matrix_vt_create(int nlin, int ncol, float* m, int rank)
{
    for(int i = 0; i < nlin; i++)
        for(int j = 0; j < ncol; j++)
            m[j + i * ncol] = i + (nlin * rank) + j * 2;
    return 0;
}

#ifdef PROFILING
void
matrix_vt_print(int nlin, int ncol, float* m)
{
    for(int i = 0; i < nlin; i++)
    {
        for(int j = 0; j < ncol; j++)
            fprintf(stderr, "%.3f ", m[j + i * ncol]);
        fprintf(stderr, "\n");
    }
}
#endif

int
main(int argc, char** argv)
{
    int         event_count = 9;
    const char* events[]    = { "infiniband:::mlx5_0_1_ext:port_xmit_data",
                             "infiniband:::mlx5_0_1_ext:port_rcv_data",
                             "infiniband:::mlx5_2_1_ext:port_xmit_data",
                             "infiniband:::mlx5_2_1_ext:port_rcv_data",
                             "infiniband:::mlx5_4_1_ext:port_xmit_data",
                             "infiniband:::mlx5_4_1_ext:port_rcv_data",
                             "infiniband:::mlx5_6_1_ext:port_xmit_data",
                             "infiniband:::mlx5_6_1_ext:port_rcv_data" };

#ifdef PAPI
    int event_set = PAPI_NULL;
    long long values[event_count];
#else
    set_papi_events(event_count, events);
#endif

    int size, rank;
    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    initialize(&argc, &argv);

    push_region("main");
    int nlin, ncol;
    if(argc < 3)
    {
        nlin = 8;
        ncol = 8;
    }
    else
    {
        nlin = atoi(argv[1]);
        ncol = atoi(argv[2]);
    }

#ifdef PAPI
    int nodes, cpn;
    if(argc < 5)
    {
        // Run the test on this many nodes.
        nodes = 2;
        // Run this many ranks per node in the test.
        cpn = 40;
    }
    else
    {
        nodes = atoi(argv[3]);
        cpn   = atoi(argv[4]);
    }
#endif

    float* matrix_C = (float*) malloc(nlin * ncol * sizeof(float));
    if(matrix_C == NULL)
    {
        fprintf(stderr, "Error in Matrix A allocation.\n");
        return 1;
    }

    // Host input and output vectors
    // Allocate memory for each vector on host
    float *vec_A, *vec_B, *vec_C;
#ifdef PINNED
    hipError_t status =
        hipHostMalloc((void**) &vec_A, (nlin / size) * ncol * sizeof(float));
    if(status != hipSuccess)
    {
        fprintf(stderr, "Error in pinned vector A allocation.\n");
        return 1;
    }
    status = hipHostMalloc((void**) &vec_B, (nlin / size) * ncol * sizeof(float));
    if(status != hipSuccess)
    {
        fprintf(stderr, "Error in pinned vector B allocation.\n");
        return 1;
    }
    status = hipHostMalloc((void**) &vec_C, (nlin / size) * ncol * sizeof(float));
    if(status != hipSuccess)
    {
        fprintf(stderr, "Error in pinned vector C allocation.\n");
        return 1;
    }
#else
    vec_A = (float*) malloc((nlin / size) * ncol * sizeof(float));
    if(vec_A == NULL)
    {
        fprintf(stderr, "Error in vector A allocation.\n");
        return 1;
    }
    vec_B = (float*) malloc((nlin / size) * ncol * sizeof(float));
    if(vec_B == NULL)
    {
        fprintf(stderr, "Error in vector B allocation.\n");
        return 1;
    }
    vec_C = (float*) malloc((nlin / size) * ncol * sizeof(float));
    if(vec_C == NULL)
    {
        fprintf(stderr, "Error in vector C allocation.\n");
        return 1;
    }
#endif

    if(matrix_vt_create(nlin / size, ncol, vec_A, rank))
    {
        fprintf(stderr, "Error in vector A creation.\n");
        return 1;
    }
    if(matrix_vt_create(nlin / size, ncol, vec_B, rank))
    {
        fprintf(stderr, "Error in vector B creation.\n");
        return 1;
    }

#ifdef PAPI
    PAPI_library_init(PAPI_VER_CURRENT);
    PAPI_create_eventset(&event_set);
    int code = 0;
    for(int i = 0; i < event_count; i++)
    {
        PAPI_event_name_to_code(events[i], &code);
        PAPI_add_event(event_set, code);
    }
#endif

    // Device input and output vectors
    float *pA, *pB, *pC;
    // Allocate memory for each vector on device
    hipMalloc((void**) &pA, ((nlin / size) * ncol) * sizeof(float));
    hipMalloc((void**) &pB, ((nlin / size) * ncol) * sizeof(float));
    hipMalloc((void**) &pC, ((nlin / size) * ncol) * sizeof(float));

    MPI_Datatype rowtype;
    MPI_Type_contiguous(ncol, MPI_FLOAT, &rowtype);
    MPI_Type_commit(&rowtype);

    int niter = 100;

    int nblocks, blockSize, taskperItem;
    taskperItem = 1;
    // Number of threads in each block
    if(argc < 6)
    {
        blockSize = 1024;
    }
    else
    {
        blockSize = min(1024, (int) (pow(2.0f, (int) ceil(log2((float) atoi(argv[5]))))));
    }
    // Number of blocks; number max is 65535
    nblocks = (int) ceil((float) ((nlin / size) * ncol) / blockSize);
    if(nblocks > 65535)
    {
        fprintf(stderr, "Number of blocks is higher than 65535!\n");
    }

#ifdef PROFILING
    double t_start = 0.0, t_end = 0.0, t = 0.0;
    t_start = MPI_Wtime();
#endif
    push_region("profiling");

#ifdef PAPI
    PAPI_start(event_set);
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    // Copy host vectors to device
    hipMemcpy(pA, vec_A, ((nlin / size) * ncol) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(pB, vec_B, ((nlin / size) * ncol) * sizeof(float), hipMemcpyHostToDevice);

    for(int i = 0; i < niter; i++)
    {
        // Execute the kernel
        MatAdd<<<nblocks, blockSize / taskperItem>>>(pA, pB, pC, (nlin / size) * ncol,
                                                     taskperItem);
    }

    // Copy array back to host
    hipMemcpy(vec_C, pC, ((nlin / size) * ncol) * sizeof(float), hipMemcpyDeviceToHost);

    // Non-collective MPI routines
    // MPI_Status recv_status;
    // MPI_Request send_status;
    // for(int r = 0; r < size; r++) {
    //   if(r != rank) {
    //      MPI_Isend(vec_C,(nlin/size),rowtype,r,rank*100+r,MPI_COMM_WORLD,&send_status);
    //      MPI_Recv(matrix_C+r*(nlin/size)*ncol,(nlin/size),rowtype,r,r*100+rank,MPI_COMM_WORLD,&recv_status);
    //      MPI_Wait(&send_status, &recv_status);
    //   }
    //}
    // for(int j = rank*(nlin/size)*ncol; j < (nlin/size)*ncol; j++) {
    //   matrix_C[j] = vec_C[j];
    //}

    // Collective MPI routine
    MPI_Allgather(vec_C, (nlin / size), rowtype, matrix_C, (nlin / size), rowtype,
                  MPI_COMM_WORLD);

#ifdef PAPI
    MPI_Barrier(MPI_COMM_WORLD);
    PAPI_stop(event_set, values);
#endif

    pop_region("profiling");

#ifdef PROFILING
    MPI_Barrier(MPI_COMM_WORLD);
    t_end = MPI_Wtime();
    t     = t_end - t_start;
#endif

#ifdef PAPI
    for(int id = 0; id < nodes; id++)
    {
        if(rank == id * cpn)
        {
            printf("\n");
            long long xmit = 0, rcv = 0;
            for(int i = 0; i < event_count; i++)
            {
                if(strstr(events[i], "xmit"))
                {
                    xmit = xmit + values[i];
                }
                else if(strstr(events[i], "rcv"))
                {
                    rcv = rcv + values[i];
                }
            }
            printf("node %d -> %lld sent bytes\n", id, xmit);
            printf("node %d -> %lld received bytes\n", id, rcv);
        }
    }
#endif

#ifdef PROFILING
    if(rank == 0)
    {
        // long double sum = 0.;
        // for(int i=0; i < nlin; i++)
        //  for(int j=0; j < ncol; j++)
        //    sum = sum + matrix_C[j+i*ncol];
        // fprintf(stderr,"Sum of all elements of the matrix: %Lf\n",sum);
        fprintf(stderr, "Time: %lf\n", t);
        // matrix_vt_print(nlin,ncol,matrix_C);
    }
#endif

    // Release host memory
    free(matrix_C);

    // Release host memory
    MPI_Type_free(&rowtype);
#ifdef PINNED
    hipHostFree(vec_A);
    hipHostFree(vec_B);
    hipHostFree(vec_C);
#else
    free(vec_A);
    free(vec_B);
    free(vec_C);
#endif

    // Release device memory
    hipFree(pA);
    hipFree(pB);
    hipFree(pC);

#ifdef PAPI
    PAPI_cleanup_eventset(event_set);
    PAPI_destroy_eventset(&event_set);
    PAPI_shutdown();
#endif

    pop_region("main");
    finalize();
    MPI_Finalize();

    return 0;
}
