#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <papi.h>

__global__ void MatAdd(float *A, float *B, float *C, int n)
{
  // Get our global thread ID
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  // Make sure we do not go out of bounds
  if(i < n)
     C[i] = A[i] + B[i];
}

int matrix_vt_create(int nlin, int ncol, float *m)
{
  for(int i=0; i < nlin; i++)
    for(int j=0; j < ncol; j++)
      m[j+i*ncol] = i + j*2;
  return 0;
}

#ifdef PROFILING
void matrix_vt_print(int nlin, int ncol, float *m)
{  
  for(int i=0; i < nlin; i++) {
    for(int j=0; j < ncol; j++)
      fprintf(stderr,"%.3f ",m[j+i*ncol]);
    fprintf(stderr,"\n");
  }
}
#endif

int main(int argc, char **argv) {

int event_set = PAPI_NULL;
int event_count = 8;
long long values[event_count]; 
const char *events[] = {
                 "infiniband:::mlx5_0_1:port_xmit_data",
		 "infiniband:::mlx5_0_1:port_rcv_data",
                 "infiniband:::mlx5_2_1:port_xmit_data",
		 "infiniband:::mlx5_2_1:port_rcv_data",
                 "infiniband:::mlx5_4_1:port_xmit_data",
		 "infiniband:::mlx5_4_1:port_rcv_data",
                 "infiniband:::mlx5_6_1:port_xmit_data",
		 "infiniband:::mlx5_6_1:port_rcv_data"
		 };

int  nlin, ncol;
if (argc < 3)
{       
   nlin = 8;
   ncol = 8;
}
else
{
   nlin = atoi(argv[1]);
   ncol = atoi(argv[2]);
}
//fprintf(stderr,"nlin(%d) ncol(%d)\n",nlin,ncol);

int nodes, cpn;
if (argc < 5)
{       
   // Run the test on this many nodes.
   nodes = 2;
   // Run this many ranks per node in the test.
   cpn = 40;
}
else
{
   nodes = atoi(argv[3]);
   cpn = atoi(argv[4]);
}
 
float *matrix_A, 
      *matrix_B;

int  size, rank;
MPI_Init(&argc, &argv);
MPI_Comm_size(MPI_COMM_WORLD, &size);
MPI_Comm_rank(MPI_COMM_WORLD, &rank);
//fprintf(stderr,"Rank(%d) Size(%d)\n",rank,size);

if(rank == 0) {
   matrix_A = (float*)malloc(nlin*ncol*sizeof(float));
   if(matrix_A == NULL) {
      fprintf(stderr,"Error in Matrix A allocation.\n");
      return 1; 
   }
   if(matrix_vt_create(nlin,ncol,matrix_A)) {
      fprintf(stderr,"Error in Matrix A creation.\n");
      return 1; 
   }

   matrix_B = (float*)malloc(nlin*ncol*sizeof(float));
   if(matrix_B == NULL) {
      fprintf(stderr,"Error in Matrix B allocation.\n");
      return 1;
   }
   if(matrix_vt_create(nlin,ncol,matrix_B)) {
      fprintf(stderr,"Error in Matrix B creation.\n");
      return 1; 
   }
}
else {
   matrix_A = NULL;
   matrix_B = NULL;
}

//Host input and output vectors
//Allocate memory for each vector on host
float *vec_A = (float*)malloc((nlin/size)*ncol*sizeof(float));
if(vec_A == NULL) {
   fprintf(stderr,"Error in vector A allocation.\n");
   return 1;
}
float *vec_B = (float*)malloc((nlin/size)*ncol*sizeof(float));
if(vec_B == NULL) {
   fprintf(stderr,"Error in vector B allocation.\n");
   return 1;
}
float *vec_C = (float*)malloc((nlin/size)*ncol*sizeof(float));
if(vec_C == NULL) {
   fprintf(stderr,"Error in vector C allocation.\n");
   return 1;
}

// Initialize PAPI
PAPI_library_init(PAPI_VER_CURRENT);
PAPI_create_eventset(&event_set);
int code = 0;
for (int i = 0; i < event_count; i++)
{
   PAPI_event_name_to_code(events[i], &code);
   PAPI_add_event(event_set, code);
}

//Device input and output vectors
float *pA, *pB, *pC;
//Allocate memory for each vector on device
hipMalloc((void**)&pA, ((nlin/size)*ncol)*sizeof(float));
hipMalloc((void**)&pB, ((nlin/size)*ncol)*sizeof(float));
hipMalloc((void**)&pC, ((nlin/size)*ncol)*sizeof(float));

MPI_Datatype rowtype;
MPI_Type_contiguous(ncol, MPI_FLOAT, &rowtype);
MPI_Type_commit(&rowtype);

//MPI_Status recv_status;
int niter = 1;

int nblocks, blockSize;
// Number of threads in each block
if(argc < 6)
{       
   blockSize = 1024;
}
else
{
   blockSize = min(1024,(int)(pow(2.0f,(int)ceil(log2((float)atoi(argv[5]))))));
}
// Number of blocks; number max is 65535
nblocks = (int)ceil((float)((nlin/size)*ncol)/blockSize);
//fprintf(stderr,"blockSize(%d) nblocks(%d)\n",blockSize,nblocks);
if(nblocks > 65535)
{
   fprintf(stderr,"Number of blocks is higher than 65535!\n");
   //return 0;
}

#ifdef PROFILING
double t_start = 0.0, t_end = 0.0, t = 0.0;
t_start = MPI_Wtime();
#endif

//StartRecordAriesCounters
PAPI_start(event_set);

MPI_Barrier(MPI_COMM_WORLD);
for(int i = 0; i < niter; i++) {
   //Initialize vectors on host
   MPI_Scatter(matrix_A,(nlin/size),rowtype,vec_A,(nlin/size),rowtype,0,MPI_COMM_WORLD);
   MPI_Scatter(matrix_B,(nlin/size),rowtype,vec_B,(nlin/size),rowtype,0,MPI_COMM_WORLD);
  
   //Copy host vectors to device
   hipMemcpy(pA, vec_A, ((nlin/size)*ncol)*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(pB, vec_B, ((nlin/size)*ncol)*sizeof(float), hipMemcpyHostToDevice);
 
   //Execute the kernel
   MatAdd<<<nblocks, blockSize>>>(pA, pB, pC, (nlin/size)*ncol);
   
   //Copy array back to host
   hipMemcpy(vec_C, pC, ((nlin/size)*ncol)*sizeof(float), hipMemcpyDeviceToHost);
   
   MPI_Gather(vec_C,(nlin/size),rowtype,matrix_A,(nlin/size),rowtype,0,MPI_COMM_WORLD);
}
MPI_Barrier(MPI_COMM_WORLD);

PAPI_stop(event_set, values);
PAPI_reset(event_set);

#ifdef PROFILING
t_end = MPI_Wtime();
t = t_end - t_start;
#endif

for(int id=0; id<nodes; id++) {
   if(rank == id * cpn) {
      printf("\n");
      long long xmit = 0, rcv = 0;
      for(int i = 0; i < event_count; i++) {
	 if(strstr(events[i],"xmit")) {
            //printf("xmit... %s: %lld\n", events[i], values[i]);
	    xmit = xmit + values[i];
	 }
	 else if(strstr(events[i],"rcv")) {
            //printf("rcv... %s: %lld\n", events[i], values[i]);
	    rcv = rcv + values[i];
	 }
      }
      printf("node %d -> %lld sent bytes\n", id, xmit);
      printf("node %d -> %lld received bytes\n", id, rcv);
   }
}

#ifdef PROFILING
if(rank == 0) {
   long double sum = 0.;
   for(int i=0; i < nlin; i++)
     for(int j=0; j < ncol; j++)
       sum = sum + matrix_A[j+i*ncol];
   fprintf(stderr,"Sum of all elements of the matrix: %Lf\n",sum);
   fprintf(stderr,"Time: %lf\n",t);
   //matrix_vt_print(nlin,ncol,matrix_A);
}
#endif

if(rank == 0) {
   fprintf(stderr,"Matrix[0][1]: %.3f\n",matrix_A[1]);
   fprintf(stderr,"Matrix[nlin-1][ncol-2]: %.3f\n",matrix_A[(ncol-2)+(nlin-1)*ncol]);
}

//Release host memory
if(rank == 0) {
   free(matrix_A);
   free(matrix_B);
}

//Release host memory
MPI_Type_free(&rowtype);
free(vec_A);
free(vec_B);
free(vec_C);

//Release device memory
hipFree(pA); 
hipFree(pB); 
hipFree(pC);

// Cleanup papi
PAPI_cleanup_eventset(event_set);
PAPI_destroy_eventset(&event_set);
PAPI_shutdown();

MPI_Finalize();
 
return 0;
}
